#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "rgb2gray.h"

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(1);
    }
}

#define TILE_W 16
#define TILE_H 16

/*******************************************************************************
 * @brief rgb2gray_cuda - CUDA kernel for converting RGB images to grayscale.
 */
__global__ void rgb2gray_cuda(uint8_t *rgb, uint8_t *gray, int W, int H)
{
    // Get thread id wrt the entire image:
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    //if ((x >= 0) && (y >= 0) && (x < W) && (y < H))
    //{
        auto p_r = rgb[y * W + x * 3 + 0];
        auto p_g = rgb[y * W + x * 3 + 1];
        auto p_b = rgb[y * W + x * 3 + 2];

        gray[y * W + x] = 0.30 * p_r + 0.59 * p_g + 0.11 * p_b;
    //}
}

/*******************************************************************************
 * @brief
 *
 */
__host__ void rgb2gray(Image &rgb, Image &gray)
{
    // TODO - Do some assertions for input parsing.
    int blockx, blocky;

    // Allocate memory for device buffers:
    uint8_t *d_in, *d_out;
    HANDLE_ERROR(hipMalloc((void **)&d_in, rgb.bytes));
    HANDLE_ERROR(hipMalloc((void **)&d_out, gray.bytes));

    // Copy data to the device buffers:
    HANDLE_ERROR(hipMemcpy(d_in, rgb.data, rgb.bytes, hipMemcpyHostToDevice));

    // Define kernel parameters:
    blockx = (rgb.W + TILE_W - 1) / TILE_W;
    blocky = (rgb.H + TILE_H - 1) / TILE_H;
    dim3 threads(TILE_W, TILE_H);
    dim3 blocks(blockx, blocky);

    // Process on the GPU
    rgb2gray_cuda<<<blocks, threads>>>(d_in, d_out, gray.W, gray.H);

    // Copy grayscale image back to host:
    HANDLE_ERROR(hipMemcpy(gray.data, d_out, gray.bytes, hipMemcpyDeviceToHost));

    hipFree(d_in);
    hipFree(d_out);
}