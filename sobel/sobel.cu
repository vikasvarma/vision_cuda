#include "hip/hip_runtime.h"
#include "sobel.h"
#include "hip/hip_runtime.h"

#define TILE_W = 16;
#define TILE_H = 16;

/*******************************************************************************
 * @brief _SOBEL - CUDA kernel for sobel edge detection filter.
 * const Image* src: Pointer to the source image.
 */
__global__ void _sobel(const Image& src, const Image& out)
{

}

Image &sobel(Image& src)
{
    return src;
}